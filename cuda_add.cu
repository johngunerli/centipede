#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_arrays(float* a, float* b, float* c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void cuda_add(float* a, float* b, float* c, int size) {
    float *d_a, *d_b, *d_c;
    
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_c, size * sizeof(float));

    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    add_arrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
